#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

/*----------------------------------------------------------------------------------------------------------------------------------------

                                             Function to get the image in order to visualize it 
                                                                
------------------------------------------------------------------------------------------------------------------------------------------*/

cv::Mat createG_x_y_Matrix(int channelId, float* gxy, int C, int R){
    
    cv::Mat gxy_cpu, tempMat;
    cv::Mat gxy_normalized, gxy_8U;
    std::vector<float> temp(3 * R * C);

    //copy all the channels from device to host
    hipMemcpy(temp.data(), gxy, 3 * R * C * sizeof(float), hipMemcpyDeviceToHost);
    
    // Choose the right channel from the temp vector
    tempMat = cv::Mat(R, C, CV_32F, temp.data() + channelId  * R * C);
    gxy_cpu = tempMat.clone();

    // Normalize the matrix to the range [0, 255] and convert to CV_8U for visualization
    cv::normalize(gxy_cpu, gxy_normalized, 0, 255, cv::NORM_MINMAX);
    gxy_normalized.convertTo(gxy_8U, CV_8U);
    
    return gxy_8U;
}


/* ----------------------------------------------------------------------------------------------------------------------------------------

                                                                CUDA Kernel for gxy calculation

------------------------------------------------------------------------------------------------------------------------------------------*/

__global__ void g_x_y_calculation(float *channel, float *gxy, int CH, int R, int CO, int nThreads){
    const int HALO_SIZE = 1; // Size of the halo for a 3x3 kernel
    __shared__ float tile[nThreads + 2*HALO_SIZE][nThreads + 2*HALO_SIZE]; // Shared memory tile with padding for Halo exchange

    // Calculate thread indices
    int tx = threadIdx.x; // Thread's x-index within the block (0 to 15)
    int ty = threadIdx.y; // Thread's y-index within the block (0 to 15)
    int z = blockIdx.z;   // Channel index

    // Calculate the top-left corner of the tile this thread will help load
    int x_start_in = blockIdx.x * nThreads - HALO_SIZE;
    int y_start_in = blockIdx.y * nThreads - HALO_SIZE;

    // Each thread loads one pixel into the shared memory tile.
    // We calculate the source coordinates in the global 'channel' buffer.
    int x_in = x_start_in + tx;
    int y_in = y_start_in + ty;

    // Cooperatively load data from global to shared memory
    // Check image boundaries before loading. If out of bounds, pad with 0.
    if (x_in >= 0 && x_in < CO && y_in >= 0 && y_in < R) {
        tile[ty][tx] = channel[z * (R * CO) + y_in * CO + x_in];
    } else {
        tile[ty][tx] = 0.0f;
    }
    // To fill the entire 18x18 tile with a 16x16 block, some threads must load more than one pixel.
    // Here, we handle the right and bottom halo edges.
    // Load right halo columns
    if (tx < 2 * HALO_SIZE) {
        x_in = x_start_in + nThreads + tx;
        if (x_in >= 0 && x_in < CO && y_in >= 0 && y_in < R) {
            tile[ty][tx + nThreads] = channel[z * (R * CO) + y_in * CO + x_in];
        } else {
            tile[ty][tx + nThreads] = 0.0f;
        }
    }
    // Load bottom halo rows
    if (ty < 2 * HALO_SIZE) {
        y_in = y_start_in + nThreads + ty;
        if (y_in >= 0 && y_in < R && x_in >= 0 && x_in < CO) {
            tile[ty + nThreads][tx] = channel[z * (R * CO) + y_in * CO + x_in];
        } else {
            tile[ty + nThreads][tx] = 0.0f;
        }
    }

    // Synchronize threads
    // This barrier ensures that the entire tile is loaded into shared memory
    // before any thread proceeds to the computation phase.
    __syncthreads();

    // Compute the convolution from shared memory
    // Calculate the global output coordinates for this thread
    int x_out = blockIdx.x * nThreads + tx;
    int y_out = blockIdx.y * nThreads + ty;

    // Ensure the output pixel is within the image bounds
    if (x_out < CO && y_out < R) {
        // Handle image borders (set to 0)
        if (x_out == 0 || x_out >= CO - 1 || y_out == 0 || y_out >= R - 1) {
            gxy[z * (R * CO) + y_out * CO + x_out] = 0.0f;
        } else {
            // The kernel weights (could be moved to __constant__ memory for another small boost)
            int W[3][3] = {{1, 2, 1}, {3, 4, 3}, {1, 2, 1}};
            float sum = 0.0f;

            // The thread's position in the shared memory tile corresponds to the center of its 3x3 window
            int shared_y = ty + HALO_SIZE;
            int shared_x = tx + HALO_SIZE;

            // Loop over the 3x3 kernel, reading from the fast shared memory tile
            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 3; j++) {
                    sum += (1.0f / 16.0f) * W[i][j] * tile[shared_y + i - HALO_SIZE][shared_x + j - HALO_SIZE];
                }
            }
            // Write the final result back to global memory
            gxy[z * (R * CO) + y_out * CO + x_out] = sum;
        }
    }
}


/* ----------------------------------------------------------------------------------------------------------------------------------------------
    
                                                                Function to process the image

    ------------------------------------------------------------------------------------------------------------------------------------------------*/



cv::Mat GetResult(std::string imagePath) {
    /* ----------------------------------------------------------------------------------------------------------------------------------------------
    
                                                                        OpenCV Setup

    ------------------------------------------------------------------------------------------------------------------------------------------------*/
    
    //Read the Image
    cv::Mat image = cv::imread(imagePath, cv::IMREAD_COLOR);

    // Decomposition of the image into its RGB channels
    if(image.empty()) {
        std::cerr << "Error: Could not open or find the image!" << std::endl;
        exit(EXIT_FAILURE);
    }  

    std::vector<cv::Mat> channels;
    cv::split(image, channels);


    /* ----------------------------------------------------------------------------------------------------------------------------------------------
    
                                                                        CUDA Setup

    ------------------------------------------------------------------------------------------------------------------------------------------------*/
    // start the timer
    auto start = std::chrono::high_resolution_clock::now();
    // get the number of threads from the environment variable
    const int nThreads = std::atoi(std::getenv("CUDA_N_THREADS"));
    std::cout << "Thread used: " << nThreads * nThreads << std::endl;

    // instatiate cv matrix from which you will get the data to be stored in CUDA memory
    std::vector<cv::Mat> ch32(3);
    std::vector<float> channel_host(3 * channels[0].rows * channels[0].cols);    

    // Convert the channels to CV_32F for CUDA processing and Flat the channels into a single vector
    for (int i = 0; i < 3; i++) {        
        // create a matrix to hold the channel data converted to CV_32F
        channels[i].convertTo(ch32[i], CV_32F);
        std::memcpy(
            channel_host.data() + i * channels[0].rows * channels[0].cols,
            ch32[i].ptr<float>(),
            channels[0].rows * channels[0].cols * sizeof(float)
        );
    }
    // Initialize the gxy vector with zeros
    std::vector<float> gxy_channels(3 * channels[0].rows * channels[0].cols, 0.0f);
    
    // Allocate memory for the channels (data as input) and gxy (data processed) on the device
    float *channel, *gxy;

    // Define the number of threads per block and the number of blocks
    dim3 threadsPerBlock(nThreads, nThreads);
    dim3 numBlocks(
        (channels[0].cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (channels[0].rows + threadsPerBlock.y - 1) / threadsPerBlock.y,
        3 // 3 channels (R, G, B)
    );
    
    // Allocate memory on the device for the channels and gxy
    hipMalloc(&channel, 3 * channels[0].rows * channels[0].cols * sizeof(float));
    hipMalloc(&gxy, 3 * channels[0].rows * channels[0].cols * sizeof(float));

    // copy the data from the image
    hipMemcpy(channel, channel_host.data(), 3 * channels[0].rows * channels[0].cols * sizeof(float), hipMemcpyHostToDevice);
    // copy the all 0s matrix that host the processed data
    hipMemcpy(gxy, gxy_channels.data(), 3 * channels[0].rows * channels[0].cols * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch the kernel to calculate gxy for each channel
    g_x_y_calculation<<<numBlocks, threadsPerBlock>>>(channel, gxy, 3, channels[0].rows, channels[0].cols, nThreads);
    

    /*-----------------------------------------------------------------------------------------------------------------------------------------------
                                                                        
                                                                    Display and Save Results

    ------------------------------------------------------------------------------------------------------------------------------------------------*/

    /* Local debugging
    cv::imshow("Red Channel", channels[2]);
    cv::imshow("Green Channel", channels[1]);
    cv::imshow("Blue Channel", channels[0]);
    cv::waitKey(0);
    */


    //                                                     Create the gxy matrices for each channel
    
    //Red Channel
    std::cout << "Red channel " << std::endl;
    cv::Mat Redgxy = createG_x_y_Matrix(2, gxy, channels[2].cols, channels[2].rows);

    //Green Channel
    std::cout << "Green channel " << std::endl;
    cv::Mat Greengxy = createG_x_y_Matrix(1, gxy, channels[1].cols, channels[1].rows);
    
    //Blue Channel
    std::cout << "Blue channel " << std::endl;
    cv::Mat Bluegxy = createG_x_y_Matrix(0, gxy, channels[0].cols, channels[0].rows);
      
    // Cuda free the memory
    hipFree(channel);
    hipFree(gxy);

    //end the timer
    auto end = std::chrono::high_resolution_clock::now();
    // Calculate the elapsed time in milliseconds
    auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "Elapsed time: " << elapsed << " ms" << std::endl;
    //Recombine the image
    cv::Mat gxyResult;
    cv::merge(std::vector<cv::Mat>{Bluegxy, Greengxy, Redgxy}, gxyResult);
    
    return gxyResult;
}

/* ----------------------------------------------------------------------------------------------------------------------------------------------
    
                                                                    Main Function

    ------------------------------------------------------------------------------------------------------------------------------------------------*/

int main(int argc, char** argv) {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        std::cout << "Dispositivo " << i << ": " << prop.name << std::endl;
        std::cout << "  Multiprocessori: " << prop.multiProcessorCount << std::endl;
        std::cout << "  Max thread per blocco: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "  Max thread per multiprocessore: " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Dimensioni massime di un blocco: "
                  << prop.maxThreadsDim[0] << " x "
                  << prop.maxThreadsDim[1] << " x "
                  << prop.maxThreadsDim[2] << std::endl;
        std::cout << "  Dimensioni massime della griglia: "
                  << prop.maxGridSize[0] << " x "
                  << prop.maxGridSize[1] << " x "
                  << prop.maxGridSize[2] << std::endl;
        std::cout << "-------------------------------\n";
    }


    for (int i = 1; i < argc; i+=2) {
        std::cout << "Processing image: " << argv[i] << std::endl;
        cv::Mat result = GetResult(argv[i]);
        std::cout << "Saving result to: " << argv[i+1] << std::endl;
        cv::imwrite(argv[i+1], result);
    }
/*
    for (const auto& subdir_entry : fs::directory_iterator(input_dir)){
        if (subdir_entry.is_directory()) {
            fs::path subdir_path = subdir_entry.path();
            for (const auto& file_entry : fs::directory_iterator(subdir_path)) {
                if (file_entry.is_regular_file() && file_entry.path().extension() == ".jpg") {
                    std::cout << "Processing image: " << file_entry.path() << std::endl;
                    cv::Mat result = GetResult(file_entry.path().string());
                    // Save the result to the output directory
                    fs::path output_path = output_dir / subdir_path.filename() / file_entry.path().filename();
                    // Create the subdirectory in the output directory if it doesn't exist
                    fs::create_directories(output_path.parent_path());
                    cv::imwrite(output_path.string(), result);
                    std::cout << "Saved result to: " << output_path << std::endl;
                }
            }
        }
    }
*/
    return 0;
}
